#include<hip/hip_runtime.h>
#include<iostream>

__global__ void simpleKernel(int a, int* dA)
{
    //this adds a value to a variable stored in global memory
    int x = threadIdx.x;
    int y = blockIdx.x;
    // printf("x is %d, y is %d, index is %d, num is %d\n",x,8*y+x,a*x+y);
    dA[8*y+x] = a*x + y;
}

int main()
{
    int hA[16], *dA;
    //allocate memory on the device (GPU); zero out all entries in this device array 
    hipMalloc((void**)&dA, sizeof(int) * 16);
    hipMemset(dA, 0, 16 * sizeof(int));

    const int RANGE = 10;
    int a = rand() % (RANGE + 1);
    //invoke GPU kernel, with one block that has four threads
    simpleKernel<<<2,8>>>(a, dA);
    hipDeviceSynchronize();
    //bring the result back from the GPU into the hostArray 
    hipMemcpy(&hA, dA, sizeof(int) * 16, hipMemcpyDeviceToHost);

    for (int i = 0; i < 16; i++)
        std::cout << hA[i] << " ";
    std::cout << "\n";
    //release the memory allocated on the GPU 
    hipFree(dA);
    return 0;
}
