#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include <chrono>
#include <random>
#include "vscale.cuh"
using std::cout;
using std::chrono::high_resolution_clock;
using std::chrono::duration;
using namespace std;



float randomWithin(float min, float max){
    int some_seed = 111;
    std::mt19937 generator(some_seed);
    std::uniform_real_distribution<float> dist(min, max);
    float pseudorandom_float = dist(generator);
    return pseudorandom_float;
}

void initializeArray(float *arr, float min, float max, unsigned int n){
    std::random_device source;
    std::mt19937_64 generator(source());
    std::uniform_real_distribution<float> dist(min, max);
    for (unsigned int i = 0; i < n;i++){
        arr[i] = dist(generator);
    }
}

void printRandFloat(float min, float max, unsigned int n){
    std::random_device source;
    std::mt19937_64 generator(source());
    std::uniform_real_distribution<float> dist(min, max);
    for (unsigned int i = 0; i < n; i++){	
	cout << dist(generator) << endl;
    }
}

   
int main(int argc, char** argv) {
    if (argc != 2) {
        return 0;
    }
    
    high_resolution_clock::time_point start;
    high_resolution_clock::time_point end;
    duration<double, std::milli> duration_sec;
    int input = atoi(argv[1]);
    unsigned int n = (unsigned int) input;
    float *a = new float[n];
    float *b = new float[n];
    float *c = new float[n];
    float *a_dev;
    float *b_dev;
    hipMalloc((void**)&a_dev, sizeof(float) * n);
    hipMalloc((void**)&b_dev, sizeof(float) * n);
    initializeArray(a, -10.0, 10.0, n);
    initializeArray(b, 0.0, 1.0, n);
    hipMemcpy(a_dev, a, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, sizeof(float)*n, hipMemcpyHostToDevice);
    const int threadsPerBlock = 16;
    int blocksPerGrid = (n+threadsPerBlock-1)/threadsPerBlock;
    
    start = high_resolution_clock::now();

    vscale<<<blocksPerGrid, threadsPerBlock>>>(a_dev, b_dev, n);
    hipDeviceSynchronize();

    end = high_resolution_clock::now();

    hipMemcpy(c, b_dev, sizeof(float) * n, hipMemcpyDeviceToHost);
    duration_sec = std::chrono::duration_cast<duration<double, std::milli>>(end - start);
    
    cout << duration_sec.count() << endl;
    cout << c[0] << endl;
    cout << c[n-1] << endl;
    delete(a);
    delete(b);
    hipFree(a_dev);
    hipFree(b_dev);
    return 0;
}
