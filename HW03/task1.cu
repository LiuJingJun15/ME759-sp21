#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>


__global__ void factorialKernel()
{
    //this adds a value to a variable stored in global memory
    int factorial = 1;
    int n = threadIdx.x+1;
    for(int i = 1; i <= n; ++i) {
        factorial *= i;
    }
    printf("%d!=%d\n", n, factorial);
}


int main()
{

    //invoke GPU kernel, with one block that has four threads
    factorialKernel<<<1,8>>>();
    hipDeviceSynchronize();
    //bring the result back from the GPU into the hostArray 
    // cudaMemcpy(&hostArray, devArray, sizeof(int) * numElems, cudaMemcpyDeviceToHost);

    // print out the result to confirm that things are looking good 
    //std::printf("here\n");    
    //release the memory allocated on the GPU 
    //cudaFree(devArray);
    return 0;
}

