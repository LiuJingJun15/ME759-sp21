#include "hip/hip_runtime.h"
#include "vscale.cuh"

__global__ void vscale(const float *a, float *b, unsigned int n)
{
    //this adds a value to a variable stored in global memory
    int x = threadIdx.x;
    int y = blockIdx.x;
    int idx = y*blockDim.x + x;
    if (idx < n){ 
        b[idx] *= a[idx];
    }else{
        return;    
    }
}
