#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include <chrono>
#include <random>
#include "reduce.cuh"
using namespace std;

float randomWithin(float min, float max){
    int some_seed = 111;
    std::mt19937 generator(some_seed);
    std::uniform_real_distribution<float> dist(min, max);
    float pseudorandom_float = dist(generator);
    return pseudorandom_float;
}

void initializeArray(float *arr, float min, float max, unsigned int n){
    std::random_device source;
    std::mt19937_64 generator(source());
    std::uniform_real_distribution<float> dist(min, max);
    for (unsigned int i = 0; i < n;i++){
        arr[i] = dist(generator);
    }
}

   
int main(int argc, char** argv) {
    if (argc != 3) {
        return 0;
    }
    int n_in = atoi(argv[1]);
    int threads_per_block = atoi(argv[2]);
    unsigned int n = (unsigned int) n_in;
    int lenReal;
    if (n % 2 == 0) {
        lenReal = n / 2;
    }
    else {
        lenReal = n / 2 + 1;
    }
    int nBlocks = (lenReal + threads_per_block - 1) / threads_per_block;
    float* input = new float[n];
    float* output = new float[nBlocks];
    initializeArray(input, -1.0, 1.0, n);
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float* input_dev;
    float* output_dev;
    
    if (nBlocks == 0) { nBlocks = 1; }
    hipMalloc((void**)&input_dev, sizeof(float) * n);
    hipMalloc((void**)&output_dev, sizeof(float) * nBlocks);
    
    hipMemcpy(input_dev, input, sizeof(float) * n, hipMemcpyHostToDevice);

    hipEventRecord(startEvent, 0);
    reduce(&input_dev, &output_dev, n, threads_per_block);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipMemcpy(output, output_dev, sizeof(float), hipMemcpyDeviceToHost);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
    cout << output[0] << endl;
    cout << elapsedTime << endl;
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    delete(input);
    delete(output);
    return 0;
}
