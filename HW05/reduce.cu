#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include "reduce.cuh"
#include<iostream>
using std::cout;
using namespace std;



__global__ void reduce_kernel(float* g_idata, float* g_odata, unsigned int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int bd = blockDim.x;
    unsigned int i = bid * (bd * 2) + tid;
    extern __shared__ float sdata[];
    // printf("bid: %d, tid:%d, bd:%d, input[i]: %f, input[i+bd]: %f\n",bid,tid,bd,g_idata[i],g_idata[i+bd]);
    if (i < n) {
        if (i + bd < n) {
            sdata[tid] = g_idata[i] + g_idata[i + bd];
        }
        else {
            sdata[tid] = g_idata[i];
        }
    }
    else {
        sdata[tid] = 0.0;
    }
    __syncthreads();
    for (unsigned int s = bd / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    g_odata[bid] = sdata[0];
    __syncthreads();
}

__host__ void reduce(float** input, float** output, unsigned int N, unsigned int threads_per_block) {
    /*
    for (int i = 0; i < N; i++) {
        printf("%f, ", (*input)[i]);
    }
    printf("\n");
    */
    int lenReal;
    if (N%2 == 0){
        lenReal = N/2;
    }else{
        lenReal = N/2 + 1;
    }
    int nBlocks = (lenReal + threads_per_block - 1) / threads_per_block;
    if (nBlocks == 0) {nBlocks = 1;}
    while (nBlocks > 0) {	
        if (nBlocks == 1){
            reduce_kernel << <1, threads_per_block, sizeof(float)*threads_per_block >> > (*input, *output, N);
            hipDeviceSynchronize();
	        break;
	    }else{
            reduce_kernel << <nBlocks, threads_per_block, sizeof(float)* threads_per_block >> > (*input, *output, N);
            hipDeviceSynchronize();
	        N = nBlocks;
            if (N % 2 == 0) {
                lenReal = N / 2;
            }
            else {
                lenReal = N / 2 + 1;
            }
            nBlocks = (lenReal + threads_per_block - 1) / threads_per_block;
	        *input = *output;            
        }
	    // printf("leaving with nBlocks: %d, lenReal: %d, N: %d\n", nBlocks, lenReal,N);
    }
}


