#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include <chrono>
#include <random>
#include "matmul.cuh"
#include "matmul_kernel.cuh"
using namespace std;

float randomFloatWithin(float min, float max){
    int some_seed = 111;
    std::mt19937 generator(some_seed);
    std::uniform_real_distribution<float> dist(min, max);
    float pseudorandom_float = dist(generator);
    return pseudorandom_float;
}

void initializeFloatArray(float *arr, float min, float max, unsigned int n){
    std::random_device source;
    std::mt19937_64 generator(source());
    std::uniform_real_distribution<float> dist(min, max);
    for (unsigned int i = 0; i < n;i++){
        arr[i] = dist(generator);
    }
}

void initializeDoubleArray(double* arr, double min, double max, unsigned int n) {
    std::random_device source;
    std::mt19937_64 generator(source());
    std::uniform_real_distribution<double> dist(min, max);
    for (unsigned int i = 0; i < n; i++) {
        arr[i] = dist(generator);
    }
}

void initializeIntArray(int* arr, int min, int max, unsigned int n) {
    int mod = max - min + 1;
    for (unsigned int i = 0; i < n; i++) {
        arr[i] = rand() % mod + min;
    }
}
   
int main(int argc, char** argv) {
    if (argc != 3) {
        return 0;
    }
    int n = atoi(argv[1]);
    int block_dim = atoi(argv[2]);
    if (block_dim > 32){
        block_dim = 32;
    }
    if (block_dim > n){
        block_dim = n;
    }

    int* A_int = new int[n * n];
    int* B_int = new int[n * n];
    int* C_int = new int[n * n];
    initializeIntArray(A_int, -10, 10, n * n);
    initializeIntArray(B_int, -10, 10, n * n);

    float* A_float = new float[n * n];
    float* B_float = new float[n * n];
    float* C_float = new float[n * n];
    initializeFloatArray(A_float, -1.0, 1.0, n * n);
    initializeFloatArray(B_float, -1.0, 1.0, n * n);
    
    double* A_double = new double[n * n];
    double* B_double = new double[n * n];
    double* C_double = new double[n * n];
    initializeDoubleArray(A_double, -1.0, 1.0, n * n);
    initializeDoubleArray(B_double, -1.0, 1.0, n * n);

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // int arrays
    int* Ad_int;
    int* Bd_int;
    int* Cd_int;
    hipMalloc((void**)&Ad_int, sizeof(int) * n * n);
    hipMalloc((void**)&Bd_int, sizeof(int) * n * n);
    hipMalloc((void**)&Cd_int, sizeof(int) * n * n);
    hipMemcpy(Ad_int, A_int, sizeof(int) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(Bd_int, B_int, sizeof(int) * n * n, hipMemcpyHostToDevice);

    hipEventRecord(startEvent, 0);
    matmul_1(Ad_int, Bd_int, Cd_int, n, block_dim);
    hipEventRecord(stopEvent, 0);

    hipEventSynchronize(stopEvent);
    hipDeviceSynchronize();
    float elapsedTime_int;
    hipEventElapsedTime(&elapsedTime_int, startEvent, stopEvent);
    hipMemcpy(C_int, Cd_int, sizeof(int) * n * n, hipMemcpyDeviceToHost);
    cout << C_int[0] << endl;
    cout << C_int[n * n - 1] << endl;
    cout << elapsedTime_int << endl;
    hipFree(Ad_int);
    hipFree(Bd_int);
    hipFree(Cd_int);
    


    // float arrays
    float* Ad_float;
    float* Bd_float;
    float* Cd_float;
    hipMalloc((void**)&Ad_float, sizeof(float) * n * n);
    hipMalloc((void**)&Bd_float, sizeof(float) * n * n);
    hipMalloc((void**)&Cd_float, sizeof(float) * n * n);
    hipMemcpy(Ad_float, A_float, sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(Bd_float, B_float, sizeof(float) * n * n, hipMemcpyHostToDevice);

    hipEventRecord(startEvent, 0);
    matmul_2(Ad_float, Bd_float, Cd_float, n, block_dim);
    hipEventRecord(stopEvent, 0);

    hipEventSynchronize(stopEvent);
    hipDeviceSynchronize();
    float elapsedTime_float;
    hipEventElapsedTime(&elapsedTime_float, startEvent, stopEvent);
    hipMemcpy(C_float, Cd_float, sizeof(float) * n * n, hipMemcpyDeviceToHost);
    cout << C_float[0] << endl;
    cout << C_float[n * n - 1] << endl;
    cout << elapsedTime_float << endl;
    hipFree(Ad_float);
    hipFree(Bd_float);
    hipFree(Cd_float);

    // double arrays
    double* Ad_double;
    double* Bd_double;
    double* Cd_double;
    hipMalloc((void**)&Ad_double, sizeof(double) * n * n);
    hipMalloc((void**)&Bd_double, sizeof(double) * n * n);
    hipMalloc((void**)&Cd_double, sizeof(double) * n * n);
    hipMemcpy(Ad_double, A_double, sizeof(double) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(Bd_double, B_double, sizeof(double) * n * n, hipMemcpyHostToDevice);

    hipEventRecord(startEvent, 0);
    matmul_3(Ad_double, Bd_double, Cd_double, n, block_dim);
    hipEventRecord(stopEvent, 0);

    hipEventSynchronize(stopEvent);
    hipDeviceSynchronize();
    float elapsedTime_double;
    hipEventElapsedTime(&elapsedTime_double, startEvent, stopEvent);
    hipMemcpy(C_double, Cd_double, sizeof(double) * n * n, hipMemcpyDeviceToHost);
    cout << C_double[0] << endl;
    cout << C_double[n * n - 1] << endl;
    cout << elapsedTime_double << endl;
    hipFree(Ad_double);
    hipFree(Bd_double);
    hipFree(Cd_double);
    return 0;
}
