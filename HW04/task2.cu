#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include <chrono>
#include <random>
#include "stencil.cuh"
using std::cout;
using std::chrono::high_resolution_clock;
using std::chrono::duration;
using namespace std;

float randomWithin(float min, float max){
    int some_seed = 111;
    std::mt19937 generator(some_seed);
    std::uniform_real_distribution<float> dist(min, max);
    float pseudorandom_float = dist(generator);
    return pseudorandom_float;
}

void initializeArray(float *arr, float min, float max, unsigned int n){
    std::random_device source;
    std::mt19937_64 generator(source());
    std::uniform_real_distribution<float> dist(min, max);
    for (unsigned int i = 0; i < n;i++){
        arr[i] = dist(generator);
    }
}

   
int main(int argc, char** argv) {
    if (argc != 4) {
        return 0;
    }
    
    int input = atoi(argv[1]);
    int R = atoi(argv[2]);
    unsigned int n = (unsigned int) input;
    int threads_per_block = atoi(argv[3]);
    float *image = new float[n];
    float *output = new float[n];
    float *mask = new float[2*R+1];
    
    initializeArray(image, -1.0, 1.0, n);
    initializeArray(mask, -1.0, 1.0, 2*R + 1);

    stencil(image, mask, output, n, R, threads_per_block); 

    delete(image);
    delete(output);
    delete(mask);
    
    return 0;
}
