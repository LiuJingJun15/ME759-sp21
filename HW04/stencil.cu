#include "hip/hip_runtime.h"
#include "stencil.cuh"
#include<iostream>
#include<hip/hip_runtime.h>
using std::cout;
using namespace std;

// The following should be stored/computed in shared memory:
// - The entire mask
// - The elements of image needed to compute the elements of output corresponding to the threads in the given block
// - The output image elements corresponding to the given block before it is written back to global memory
__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R){
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int BLOCK_SIZE = blockDim.x;
    int index = bx*BLOCK_SIZE + tx;
    extern __shared__ float sh_arr[];
    // initialize shared mem
        for (int i = 0; i < 2*R+1; i++){
	    sh_arr[i] = mask[i];
	}
    int offsetImg = 2*R+1;
    if (tx - R < 0){
        if (BLOCK_SIZE*bx - R < 0){
	    sh_arr[offsetImg + tx] = 1.0;
	}else{
	    sh_arr[offsetImg + tx] = image[index - R];
	}
	sh_arr[offsetImg + tx + R] = image[index];
    }else if (tx + R > BLOCK_SIZE - 1){
        if (BLOCK_SIZE*bx + tx + R > n-1){
	    sh_arr[offsetImg + 2*R + tx] = 1.0;
	}else{
	    sh_arr[offsetImg + 2*R + tx] = image[index + R];
	}
	sh_arr[offsetImg + tx + R] = image[index];
    }else{
	sh_arr[offsetImg + tx + R] = image[index];
    }
    __syncthreads();
    // compute
    int offsetOut = offsetImg + BLOCK_SIZE + 2*R;
    for (int i=0; i < 2*R+1; i++){
        sh_arr[offsetOut + tx] += sh_arr[i] * sh_arr[offsetImg + i + tx];
    }
    __syncthreads();
    output[index] = sh_arr[offsetOut + tx];
}

__host__ void stencil(const float* image,
                      const float* mask,
                      float* output,
                      unsigned int n,
                      unsigned int R,
                      unsigned int threads_per_block){
                          hipEvent_t startEvent, stopEvent; 
                          hipEventCreate(&startEvent); 
                          hipEventCreate(&stopEvent);
                          float *image_dev;
                          float *output_dev;
                          float *mask_dev;
                          hipMalloc((void**)&image_dev, sizeof(float) * n );
                          hipMalloc((void**)&output_dev, sizeof(float) * n);
                          hipMalloc((void**)&mask_dev, sizeof(float) *(2 * R + 1));
                          hipMemcpy(image_dev, image, sizeof(float)*n, hipMemcpyHostToDevice);
                          hipMemcpy(mask_dev, mask, sizeof(float)*(2*R+1), hipMemcpyHostToDevice);

                          int blocksPerGrid = (n + threads_per_block - 1)/threads_per_block;
	                      int sizeShared = sizeof(float)*(2*threads_per_block + 4*R+1); //2R + 1 for image, 2R+1 for mask, block_size for output
                          hipEventRecord(startEvent, 0);

                          stencil_kernel<<<blocksPerGrid, threads_per_block, sizeShared>>>(image_dev, mask_dev, output_dev, n, R);
			  hipDeviceSynchronize();
                          hipEventRecord(stopEvent, 0); 

                          hipEventSynchronize(stopEvent); 

                          float elapsedTime; 
                          hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
                          hipMemcpy(output, output_dev, sizeof(float) * n, hipMemcpyDeviceToHost);
                          cout << output[n-1] << endl;
                          cout << elapsedTime << endl;
                          hipFree(image_dev);
                          hipFree(output_dev);
                          hipFree(mask_dev);
                          hipEventDestroy(startEvent);
                          hipEventDestroy(stopEvent);
                      }
