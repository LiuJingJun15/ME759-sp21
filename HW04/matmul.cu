#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include<iostream>
#include<hip/hip_runtime.h>
using std::cout;
using namespace std;
void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block){
    hipEvent_t startEvent, stopEvent; 
    hipEventCreate(&startEvent); 
    hipEventCreate(&stopEvent);
    int blocksPerGrid = (n * n + threads_per_block-1)/threads_per_block;
    float *a_dev;
    float *b_dev;
    float *c_dev;
    hipMalloc((void**)&a_dev, sizeof(float) * n * n);
    hipMalloc((void**)&b_dev, sizeof(float) * n * n);
    hipMalloc((void**)&c_dev, sizeof(float) * n * n);
    hipMemcpy(a_dev, A, sizeof(float)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, B, sizeof(float)*n*n, hipMemcpyHostToDevice);

    hipEventRecord(startEvent, 0);
    matmul_kernel<<<blocksPerGrid, threads_per_block>>>(a_dev, b_dev, c_dev, n);
    hipDeviceSynchronize();
    hipEventRecord(stopEvent, 0); 
    hipEventSynchronize(stopEvent); 

    float elapsedTime; 
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
    hipMemcpy(C, c_dev, sizeof(float) * n * n, hipMemcpyDeviceToHost);
    cout << C[n*n-1] << endl;
    cout << elapsedTime << endl;
    
    hipEventDestroy(startEvent); 
    hipEventDestroy(stopEvent);
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
}

__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n){
    int bidx = blockIdx.x;
    int tidx = threadIdx.x;
    int M = blockDim.x;
    int idxC = M*bidx + tidx;
    int row = idxC/n;
    int col = idxC%n;
    float sum = 0.0;
    if (idxC >= n*n){
        return;
    }
    for (int i = 0; i < n; i++){
        sum += A[row*n + i]*B[n*i + col];
    }
    C[idxC] = sum;
}
