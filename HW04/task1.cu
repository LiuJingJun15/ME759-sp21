#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include <chrono>
#include <random>
#include "matmul.cuh"
using std::cout;
using std::chrono::high_resolution_clock;
using std::chrono::duration;
using namespace std;

float randomWithin(float min, float max){
    int some_seed = 111;
    std::mt19937 generator(some_seed);
    std::uniform_real_distribution<float> dist(min, max);
    float pseudorandom_float = dist(generator);
    return pseudorandom_float;
}

void initializeArray(float *arr, float min, float max, unsigned int n){
    std::random_device source;
    std::mt19937_64 generator(source());
    std::uniform_real_distribution<float> dist(min, max);
    for (unsigned int i = 0; i < n;i++){
        arr[i] = dist(generator);
    }
}

   
int main(int argc, char** argv) {
    if (argc != 3) {
        return 0;
    }
    int input = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);
    unsigned int n = (unsigned int) input;
    float *a = new float[n*n];
    float *b = new float[n*n];
    float *c = new float[n*n];
    initializeArray(a, -1.0, 1.0, n * n);
    initializeArray(b, -1.0, 1.0, n * n);

    matmul(a, b, c, n, threadsPerBlock); 

    delete(a);
    delete(b);
    delete(c);
    
    return 0;
}
