#include<hip/hip_runtime.h>
#include<iostream>
#include <chrono>
#include <random>
#include <thrust/reduce.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/system/omp/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/sort.h>
#include <thrust/inner_product.h>

#include "count.cuh"

typedef thrust::tuple<int, int>       Tuple2;
struct find_occurence {
    __host__ __device__
        Tuple2 operator()(Tuple2 t) {
        int x, y; thrust::tie(x, y) = t;
        int u = x;
        int v = y/x;
        return Tuple2(u, v);
    }
};


void count(const thrust::device_vector<int>& d_in,
    thrust::device_vector<int>& values,
    thrust::device_vector<int>& counts) {

    int n = d_in.end() - d_in.begin();
    thrust::device_vector<int> d_in_copy(n);
    thrust::copy(thrust::device, d_in.begin(), d_in.end(), d_in_copy.begin());

    thrust::sort(thrust::device, d_in_copy.begin(), d_in_copy.end());

    int sizeval = thrust::inner_product(d_in_copy.begin(), d_in_copy.end() - 1,
        d_in_copy.begin() + 1,
        0,
        thrust::plus<int>(),
        thrust::not_equal_to<int>()) + 1;

    values.resize(sizeval);

    thrust::fill(counts.begin(), counts.end(), 1);
    thrust::reduce_by_key(d_in_copy.begin(), d_in_copy.end(), counts.begin(), values.begin(), counts.begin());

    counts.resize(sizeval);
    /*
    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(values.begin(), counts.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(values.end(), counts.end())),
        thrust::make_zip_iterator(thrust::make_tuple(values.begin(), counts.begin())),
        find_occurence());
    
    for (int i = 0; i < sizeval; i++){
         counts[i] = counts[i]/values[i];
    }
    */
}
