#include<hip/hip_runtime.h>
#include<iostream>
#include <chrono>
#include <random>
#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <stdio.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include "cub/util_debug.cuh"
using namespace std;
using namespace hipcub;
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


int main(int argc, char** argv) {
    if (argc != 2) {
        return 0;
    }
    int n = atoi(argv[1]);
    const size_t num_items = n;
    float *h_in = new float[num_items];
    std::random_device source;
    std::mt19937_64 generator(source());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    for (int i = 0; i < n; i++) {
        h_in[i] = dist(generator);
    }
    
    
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    float* d_in = NULL;
    g_allocator.DeviceAllocate((void**)&d_in, sizeof(float) * num_items);
    hipMemcpy(d_in, h_in, sizeof(float) * num_items, hipMemcpyHostToDevice);
    float* d_sum = NULL;
    g_allocator.DeviceAllocate((void**)&d_sum, sizeof(float) * 1);
    // Request and allocate temporary storage
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, num_items);
    g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes);
    
    hipEventRecord(startEvent, 0);

    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, num_items);

    hipEventRecord(stopEvent, 0);

    float gpu_sum;
    hipMemcpy(&gpu_sum, d_sum, sizeof(float) * 1, hipMemcpyDeviceToHost);
    hipEventSynchronize(stopEvent);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
    cout << gpu_sum << endl;
    cout << elapsedTime << endl;
    if (d_in) g_allocator.DeviceFree(d_in);
    if (d_sum) g_allocator.DeviceFree(d_sum);
    if (d_temp_storage) g_allocator.DeviceFree(d_temp_storage);
    return 0;
}
