#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include <chrono>
#include <random>
#include <thrust/reduce.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/system/omp/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/sequence.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_real_distribution.h>

using namespace std;

struct randg
{
    __host__ __device__
        float operator()(const float min, const float max) const{
        thrust::minstd_rand rng;
        thrust::uniform_real_distribution<float> dist(min, max);

        return dist(rng);
    }
};

int main(int argc, char** argv) {
    if (argc != 2) {
        return 0;
    }
    int n = atoi(argv[1]);
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    thrust::host_vector<float> h_vec(n);
    thrust::device_vector<float> d_vec;

    thrust::minstd_rand rng;
    thrust::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    for (int i = 0; i < n; i++){
        h_vec[i] = dist(rng);
    }
    float init = h_vec[0];
    // thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());
    d_vec = h_vec;
    hipEventRecord(startEvent, 0);

    float result = thrust::reduce(thrust::cuda::par, d_vec.begin(), d_vec.end(), init);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
    cout << result << endl;
    cout << elapsedTime << endl;
    return 0;
}
