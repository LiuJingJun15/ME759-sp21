#include<hip/hip_runtime.h>
#include<iostream>
#include <chrono>
#include <random>
#include <thrust/reduce.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/system/omp/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/sequence.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include "count.cuh"
#include <ctime>
using namespace std;

int main(int argc, char** argv) {
    if (argc != 2) {
        return 0;
    }
    int n = atoi(argv[1]);
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    thrust::host_vector<int> h_vec(n);
    thrust::device_vector<int> d_val(n);
    thrust::device_vector<int> d_count(n);
    thrust::device_vector<int> d_in(n);
    thrust::host_vector<int> value(n);
    thrust::host_vector<int> counts(n);
    srand(time(0));
    for (int i = 0; i < n; i++) {
        h_vec[i] = rand()%501;
    }
    d_in = h_vec;
    hipEventRecord(startEvent, 0);

    count(d_in, d_val, d_count);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    int sizeval = d_val.end() - d_val.begin();
    value = d_val;
    counts = d_count;
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
    
    cout << value[sizeval - 1] << endl;
    cout << counts[sizeval - 1] << endl;
    cout << elapsedTime << endl;
    
}
